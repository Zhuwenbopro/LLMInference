#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <chrono>

__global__ void silu_fp16_kernel(const __half* input, __half* output, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < size) {
        __half x_half = input[idx];
        // 将 __half 转换为 float 进行计算
        float x = __half2float(x_half);
        // 计算 sigmoid(x) = 1 / (1 + exp(-x))
        float sigmoid = 1.0f / (1.0f + expf(-x));
        // 计算 SiLU(x) = x * sigmoid(x)
        float silu = x * sigmoid;
        // 将结果转换回 __half
        output[idx] = __float2half(silu);
    }
}


__global__ void silu_fp32_kernel(const float* input, float* output, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        // 计算 sigmoid(x)
        float sigmoid = 1.0f / (1.0f + expf(-x));
        // 计算 x * sigmoid(x)
        output[idx] = x * sigmoid;
    }
}

int main() {
    // 定义数据大小
    const int size = 1 << 24; // 大约 16M 个元素
    const int bytes_fp16 = size * sizeof(__half);
    const int bytes_fp32 = size * sizeof(float);

    // 分配主机内存
    float* h_input_fp32 = new float[size];
    float* h_output_fp32 = new float[size];
    float* h_output_fp16 = new float[size];

    // 初始化输入数据
    for (int i = 0; i < size; ++i) {
        h_input_fp32[i] = static_cast<float>(rand()) / RAND_MAX; // 随机数在 0 到 1 之间
    }

    // 分配设备内存
    float* d_input_fp32;
    float* d_output_fp32;
    __half* d_input_fp16;
    __half* d_output_fp16;

    hipMalloc(&d_input_fp32, bytes_fp32);
    hipMalloc(&d_output_fp32, bytes_fp32);
    hipMalloc(&d_input_fp16, bytes_fp16);
    hipMalloc(&d_output_fp16, bytes_fp16);

    // 将输入数据复制到设备（FP32）
    hipMemcpy(d_input_fp32, h_input_fp32, bytes_fp32, hipMemcpyHostToDevice);

    // 将 FP32 输入数据转换为 FP16 并复制到设备
    __half* h_input_fp16 = new __half[size];
    for (int i = 0; i < size; ++i) {
        h_input_fp16[i] = __float2half(h_input_fp32[i]);
    }
    hipMemcpy(d_input_fp16, h_input_fp16, bytes_fp16, hipMemcpyHostToDevice);

    // 定义 CUDA 内核执行配置
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    // 计时变量
    hipEvent_t start_fp32, stop_fp32;
    hipEvent_t start_fp16, stop_fp16;
    hipEventCreate(&start_fp32);
    hipEventCreate(&stop_fp32);
    hipEventCreate(&start_fp16);
    hipEventCreate(&stop_fp16);

    // 运行并计时 FP32 核函数
    hipEventRecord(start_fp32);
    silu_fp32_kernel<<<blocks, threads>>>(d_input_fp32, d_output_fp32, size);
    hipEventRecord(stop_fp32);

    // 运行并计时 FP16 核函数
    hipEventRecord(start_fp16);
    silu_fp16_kernel<<<blocks, threads>>>(d_input_fp16, d_output_fp16, size);
    hipEventRecord(stop_fp16);

    // 等待计算完成
    hipDeviceSynchronize();

    // 计算 FP32 运行时间
    float time_fp32 = 0;
    hipEventElapsedTime(&time_fp32, start_fp32, stop_fp32);

    // 计算 FP16 运行时间
    float time_fp16 = 0;
    hipEventElapsedTime(&time_fp16, start_fp16, stop_fp16);

    // 将结果复制回主机（FP32）
    hipMemcpy(h_output_fp32, d_output_fp32, bytes_fp32, hipMemcpyDeviceToHost);

    // 将结果复制回主机（FP16），并转换为 FP32 以便比较
    hipMemcpy(h_input_fp16, d_output_fp16, bytes_fp16, hipMemcpyDeviceToHost);
    for (int i = 0; i < size; ++i) {
        h_output_fp16[i] = __half2float(h_input_fp16[i]);
    }

    // 输出运行时间
    std::cout << "FP32 kernel execution time: " << time_fp32 << " ms" << std::endl;
    std::cout << "FP16 kernel execution time: " << time_fp16 << " ms" << std::endl;

    // 验证结果的正确性（可选）
    int error_count = 0;
    for (int i = 0; i < size; ++i) {
        float diff = fabs(h_output_fp32[i] - h_output_fp16[i]);
        if (diff > 1e-2) { // 允许一定的误差
            error_count++;
        }
    }
    if (error_count == 0) {
        std::cout << "Results are correct within acceptable error margin." << std::endl;
    } else {
        std::cout << "There are " << error_count << " mismatches between FP32 and FP16 results." << std::endl;
    }

    // 释放资源
    delete[] h_input_fp32;
    delete[] h_input_fp16;
    delete[] h_output_fp32;
    delete[] h_output_fp16;
    hipFree(d_input_fp32);
    hipFree(d_output_fp32);
    hipFree(d_input_fp16);
    hipFree(d_output_fp16);
    hipEventDestroy(start_fp32);
    hipEventDestroy(stop_fp32);
    hipEventDestroy(start_fp16);
    hipEventDestroy(stop_fp16);

    return 0;
}
