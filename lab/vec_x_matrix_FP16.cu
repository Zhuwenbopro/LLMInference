#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

// 错误检查宏
#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__      \
                      << " - " << hipGetErrorString(err) << std::endl;       \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

#define CHECK_CUBLAS(call)                                                    \
    do {                                                                      \
        hipblasStatus_t err = call;                                            \
        if (err != HIPBLAS_STATUS_SUCCESS) {                                   \
            std::cerr << "cuBLAS Error at " << __FILE__ << ":" << __LINE__    \
                      << " - " << err << std::endl;                           \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

int main() {
    // 矩阵和向量的尺寸
    int M = 4096; // 向量长度
    int N = 4096; // 矩阵列数

    // 分配主机内存
    size_t size_vector = M * sizeof(__half);
    size_t size_matrix = M * N * sizeof(__half);
    size_t size_result = N * sizeof(__half);

    __half* h_x = (__half*)malloc(size_vector);   // 向量 x
    __half* h_A = (__half*)malloc(size_matrix);   // 矩阵 A
    __half* h_y = (__half*)malloc(size_result);   // 结果向量 y

    // 初始化向量 x 和矩阵 A
    for (int i = 0; i < M; ++i) {
        h_x[i] = __float2half(1.0f); // 或者使用随机值
    }
    for (int i = 0; i < M * N; ++i) {
        h_A[i] = __float2half(1.0f); // 或者使用随机值
    }

    // 分配设备内存
    __half* d_x;
    __half* d_A;
    __half* d_y;
    CHECK_CUDA(hipMalloc((void**)&d_x, size_vector));
    CHECK_CUDA(hipMalloc((void**)&d_A, size_matrix));
    CHECK_CUDA(hipMalloc((void**)&d_y, size_result));

    // 将数据从主机复制到设备
    CHECK_CUDA(hipMemcpy(d_x, h_x, size_vector, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_A, h_A, size_matrix, hipMemcpyHostToDevice));

    // 创建 cuBLAS 句柄
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // 设置数学模式以允许使用 Tensor Core
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    // 设置 CUDA 事件，用于计时
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    float milliseconds = 0.0f;

    // 执行向量乘以矩阵操作：y = alpha * x * A + beta * y
    // 在 cuBLAS 中，向量和矩阵需要以列主序存储，因此需要调整参数
    float alpha = 1.0f;
    float beta = 0.0f;

    // 开始计时
    CHECK_CUDA(hipEventRecord(start));

    // 使用 cublasGemmEx 实现 y = x * A
    CHECK_CUBLAS(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, 1, M,
        &alpha,
        d_A, HIP_R_16F, N,
        d_x, HIP_R_16F, M,
        &beta,
        d_y, HIP_R_16F, N,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // 停止计时
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "运行时间: " << milliseconds << " ms" << std::endl;

    // 将结果从设备复制回主机
    CHECK_CUDA(hipMemcpy(h_y, d_y, size_result, hipMemcpyDeviceToHost));

    // 验证结果（简单验证）
    // 由于 x 和 A 的元素都是 1，预期 y 的每个元素都是 M
    bool correct = true;
    for (int i = 0; i < N; ++i) {
        float y_value = __half2float(h_y[i]);
        if (fabs(y_value - M) > 1e-2) {
            correct = false;
            std::cerr << "结果错误，索引 " << i << "，值: " << y_value << std::endl;
            break;
        }
    }
    if (correct) {
        std::cout << "结果正确！" << std::endl;
    } else {
        std::cout << "结果错误！" << std::endl;
    }

    // 清理资源
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_y));
    free(h_x);
    free(h_A);
    free(h_y);
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return 0;
}
